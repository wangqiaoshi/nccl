/*************************************************************************
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

// Based on //sw/gpgpu/MachineLearning/cudnn/test/testUtil.cpp

#define POLYNOMIAL 0x04c11db7L      // Standard CRC-32 ppolynomial
static unsigned int crc_table[256]; // Table of 8-bit remainders
static int tableLoaded = 0;

static void crcInit(void) {
  int i, j;
  unsigned int crc_accum;

  for (i=0;  i<256;  i++) {
    crc_accum = ( i << 24 );
    for ( j = 0;  j < 8;  j++ ) {
      if ( crc_accum & 0x80000000L )
        crc_accum = (crc_accum << 1) ^ POLYNOMIAL;
      else
        crc_accum = (crc_accum << 1);
    }
    crc_table[i] = crc_accum;
  }
}

unsigned calcCRCHost(unsigned char *data_blk_ptr, size_t data_blk_size) {
  if (tableLoaded == 0) {
    crcInit();
    tableLoaded = 1;
  }

  unsigned int crc_accum = 0x11223344; // Initial CRC value used in cuDNN
  int i;
  for (size_t j=0; j<data_blk_size; j++) {
    i = ((int) (crc_accum >> 24) ^ *data_blk_ptr++) & 0xFF;
    crc_accum = (crc_accum << 8) ^ crc_table[i];
  }
  crc_accum = ~crc_accum;
  return crc_accum;
}


static __global__ void CRCKernel(unsigned char* data, int bytes, int rank) {
  __shared__ unsigned crc_table[256];
  __shared__ unsigned char buffer[256];

  // Build table of 8-bit remainders
  int crc_accum = threadIdx.x << 24;
  for (int j=0; j<8; ++j) {
    const int mask = (crc_accum & 0x80000000) ? POLYNOMIAL : 0;
    crc_accum = (crc_accum << 1) ^ mask;
  }
  crc_table[threadIdx.x] = crc_accum;

  unsigned int crc_val = 0x11223344; // Initial CRC value used in cuDNN
  for(int i=threadIdx.x; i<bytes; i+=256) {
    buffer[threadIdx.x] = data[i];
    __syncthreads();

    if (threadIdx.x == 0) {
      const int remaining = bytes - i;
      const int n = (remaining > 256) ? 256 : remaining;
      for(int j=0; j<n; ++j) {
        int t = ((int)(crc_val >> 24) ^ buffer[j]) & 0xFF;
        crc_val = (crc_val << 8) ^ crc_table[t];
      }
    }
    __syncthreads();
  }

  if (threadIdx.x == 0)
    printf("NCCL Rank %d CRC 0x%.8x\n", rank, ~crc_val);
}

void printCRCDev(unsigned char* data,
                 int bytes,
                 int rank,
                 hipStream_t stream)
{
  CRCKernel<<<1, 256, 0, stream>>>(data, bytes, rank);
}
